#include "hip/hip_runtime.h"
#include <iostream>
#include <complex>
#include <vector>
#include <chrono>

#include <cuda/std/complex>
#include <thrust/system_error.h>
#include <thrust/system/cuda/error.h>

#include "window.h"
#include "save_image.h"
#include "utils.h"

// Use an alias to simplify the use of complex type
using Complex = cuda::std::complex<float>;

#define cuda_err_chk(ans) { cuda_throw((ans), __FILE__, __LINE__); }

inline void cuda_throw(hipError_t code, const char *file, int line) {
    if (code != hipSuccess) {
        std::stringstream ss;
        ss << file << "(" << line << ")";
        auto file_and_line = ss.str();
        throw thrust::system_error(code, thrust::cuda_category(), file_and_line);
    }
}

// Convert a pixel coordinate to the complex domain
__device__
Complex scale(const window<int> &scr, const window<float> &fr, Complex c) {
    Complex aux(c.real() / (float) scr.width() * fr.width() + fr.x_min(),
                c.imag() / (float) scr.height() * fr.height() + fr.y_min());
    return aux;
}

// Check if a point is in the set or escapes to infinity, return the number if iterations
__device__
int escape_mandelbrot(Complex c, int iter_max) {
    Complex z(0);
    int iter = 0;

    while (cuda::std::abs(z) < 2.0 && iter < iter_max) {
        z = z * z + c;
        ++iter;
    }

    return iter;
}

// Loop over each pixel from our image and check if the points associated with this pixel escape to infinity
__global__
void get_number_iterations(const window<int> *scr, const window<float> *fract, int iter_max, int *colors) {
    int row_idx = blockIdx.y * blockDim.y + threadIdx.y;
    int col_idx = blockIdx.x * blockDim.x + threadIdx.x;
    int row_stride = blockDim.y * gridDim.y;
    int col_stride = blockDim.x * gridDim.x;

    for (int i = row_idx; i < scr->height(); i += row_stride) {
        for (int j = col_idx; j < scr->width(); j += col_stride) {
            Complex c((float) (scr->x_min() + j), (float) (scr->y_min() + i));
            c = scale(*scr, *fract, c);
            colors[row_idx * scr->width() + col_idx] = escape_mandelbrot(c, iter_max);
        }
    }
}

void fractal(window<int> &scr, window<float> &fract, int iter_max, const char *fname, bool smooth_color) {
    auto start = std::chrono::steady_clock::now();
    window<int> *scr_uni;
    window<float> *fract_uni;
    int *colors_gpu;

    cuda_err_chk(hipMallocManaged(&scr_uni, sizeof(window<int>)));
    cuda_err_chk(hipMallocManaged(&fract_uni, sizeof(window<float>)));
    cuda_err_chk(hipMalloc(&colors_gpu, sizeof(int) * scr.size()));
    *scr_uni = scr;
    *fract_uni = fract;

    dim3 threads_per_block(16, 16);
    dim3 n_blocks((scr.width() + threads_per_block.x - 1) / threads_per_block.x,
                  (scr.height() + threads_per_block.y - 1) / threads_per_block.y);
    get_number_iterations<<<n_blocks, threads_per_block>>>(scr_uni, fract_uni, iter_max, colors_gpu);
    cuda_err_chk(hipGetLastError());
    cuda_err_chk(hipDeviceSynchronize());

    std::vector<int> colors(scr.size());
    cuda_err_chk(hipMemcpy(colors.data(), colors_gpu, sizeof(int) * scr.size(), hipMemcpyDeviceToHost));

    auto end = std::chrono::steady_clock::now();
    std::cout << "Time to generate " << fname << " = " << std::chrono::duration<float, std::milli>(end - start).count()
              << " [ms]" << std::endl;

    cuda_err_chk(hipFree(scr_uni));
    cuda_err_chk(hipFree(fract_uni));
    cuda_err_chk(hipFree(colors_gpu));

    // Save (show) the result as an image
    plot(scr, colors, iter_max, fname, smooth_color);
}

void mandelbrot() {
    // Define the size of the image
    window<int> scr(0, 2400, 0, 2400);
    // The domain in which we test for points
    window<float> fract(-2.2, 1.2, -1.7, 1.7);

    int iter_max = 500;
    const char *fname = "mandelbrot.png";
    bool smooth_color = true;

    // Experimental zoom (bugs ?). This will modify the fract window (the domain in which we calculate the fractal function)
    //zoom(1.0, -1.225, -1.22, 0.15, 0.16, fract); //Z2
    fractal(scr, fract, iter_max, fname, smooth_color);
}

int main() {
    mandelbrot();
    return 0;
}
